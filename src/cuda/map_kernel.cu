#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

template <typename T>
__device__ auto map_offsets_in_place(T *data, const std::size_t *idx_to_offsets,
                                     std::size_t size) -> void {
  auto tid = threadIdx.x;
  auto idx = blockIdx.x * blockDim.x + threadIdx.x; // # in grid.

  if (idx >= size) {
    return;
  }

  auto tensor_idx = idx_to_offsets[idx]; // offset in data.
  auto input = data[tensor_idx];

  auto group = cooperative_groups::this_grid();

  group.sync();

  data[idx] = input;
}

#define EXTERN(T, SUFFIX)                                                      \
  extern "C" __global__ void map_offsets_##SUFFIX(                             \
      T *in, const std::size_t *offsets, std::size_t size) {                   \
    map_offsets_in_place(in, offsets, size);                                   \
  }

// actually they are not always f32/f64 by cpp standard but for simplicity -
// assume that yes.
EXTERN(float, f32) // currently lets keep it and make it f32 only.
